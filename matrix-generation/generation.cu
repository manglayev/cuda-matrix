#include "hip/hip_runtime.h"
#include "generation_header.cuh"
#include ""
//#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

__global__ void cuda_global(int *dev_a)
{
  if (threadIdx.x < 1)
  {
    dev_a[0] = 0;
  }
}

std::array<int, N> generateArray()
{
  std::default_random_engine engine{};
  std::uniform_int_distribution randomNumbers{-9, 9};
  std::array<int, N> a{};
  for(int i = 0; i < N; i++)
  {
    a[i] = randomNumbers(engine);
  }
  return a;
}
/*
print N x N matrix
void printArray(std::array<std::array<int, N>, N>& a)
{
  // loop through array's rows
  for (const auto& row : a)
  {
    // loop through columns of current row
    for (const auto& element : row)
    {
      std::cout << element << ' ';
    }
    std::cout << '\n'; // start new line of output
  }
}
*/
void wrapper()
{
  printf("STAGE 3 WRAPPER START\n");
  std::array a{generateArray()};
  
  for (const auto& element : a)
  {
      std::cout << element << ' ';
  }
  std::cout << '\n'; // start new line of output
  
  //printArray(a);
  /*
  int *dev_a;
  hipMalloc((void**)&dev_a, CUDASIZE*sizeof(int));
  hipMemcpy(dev_a, a, CUDASIZE*sizeof(int), hipMemcpyHostToDevice);
  cuda_global<<<BLOCKS, THREADS>>>(dev_a);
  hipMemcpy(a, dev_a, sizeof(int), hipMemcpyDeviceToHost);
  hipFree(dev_a);
*/
  printf("STAGE 3 WRAPPER END\n");
}
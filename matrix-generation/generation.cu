#include "hip/hip_runtime.h"
#include "generation_header.cuh"
#include ""
//#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

__global__ void cuda_global(int *dev_a)
{
  if (threadIdx.x < 1)
  {
    dev_a[0] = 0;
  }
}
//generate N x N array
std::array<std::array<int, N>, N> generateArray()
{
  std::default_random_engine engine{};
  std::uniform_int_distribution randomNumbers{-9, 9};
  std::array<std::array<int, N>, N> a{};
  for(int i = 0; i < N; i++)
  {
    for(int j = 0; j < N; j++)
    {
      a[i][j] = randomNumbers(engine);
    }
  }
  return a;
}


//print N x N matrix
void printArray(std::array<std::array<int, N>, N>& a)
{
  // loop through array's rows
  for (const auto& row : a)
  {
    // loop through columns of current row
    for (const auto& element : row)
    {
      if(element >= 0)
        std::cout << ' ' << element << ' ';
      else
        std::cout << element << ' ';
    }
    std::cout << '\n'; // start new line of output
  }
}

void wrapper()
{
  printf("STAGE 3 WRAPPER START\n");
  std::array<std::array<int, N>, N> a = generateArray();
  printArray(a);
  /*
  int *dev_a;
  hipMalloc((void**)&dev_a, CUDASIZE*sizeof(int));
  hipMemcpy(dev_a, a, CUDASIZE*sizeof(int), hipMemcpyHostToDevice);
  cuda_global<<<BLOCKS, THREADS>>>(dev_a);
  hipMemcpy(a, dev_a, sizeof(int), hipMemcpyDeviceToHost);
  hipFree(dev_a);
*/
  printf("STAGE 3 WRAPPER END\n");
}